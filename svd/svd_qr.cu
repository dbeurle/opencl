
// C++-ified SVD example from
// https://docs.nvidia.com/cuda/cusolver/index.html#svd_examples

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include <cassert>
#include <iostream>
#include <vector>

inline void printMatrix(int m, int n, const double* A, int lda, const char* name)
{
    for (int row = 0; row < m; row++)
    {
        for (int col = 0; col < n; col++)
        {
            std::cout << name << "(" << row + 1 << ", " << col + 1 << ") = " << A[row + col * lda]
                      << '\n';
        }
    }
}

inline void check(hipError_t&& error)
{
    if (error != hipSuccess)
    {
        throw std::domain_error("CUDA error");
    }
}

inline void check(hipblasStatus_t&& status)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        throw std::domain_error("Cuda BLAS error");
    }
}

inline void check(hipsolverStatus_t&& status)
{
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        throw std::domain_error("Cuda BLAS error");
    }
}

int main(int argc, char* argv[])
{
    constexpr int rows = 3;
    constexpr int cols = 2;
    constexpr int lda = rows;

    //       | 1 2  |
    //   A = | 4 5  |
    //       | 2 1  |
    std::vector<double> A = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0};

    int info_gpu = 0;

    std::cout << "A = (matlab base-1)\n";
    printMatrix(rows, cols, A.data(), lda, "A");
    std::cout << "=====\n";

    // step 1: create cusolverDn/cublas handle
    hipsolverHandle_t cusolver_handle = nullptr;
    hipblasHandle_t cublas_handle = nullptr;

    check(hipsolverDnCreate(&cusolver_handle));
    check(hipblasCreate(&cublas_handle));

    // step 2: copy A and B to device
    double* d_A = nullptr;
    double* d_S = nullptr;
    double* d_U = nullptr;
    double* d_VT = nullptr;
    int* devInfo = nullptr;
    double* d_W = nullptr; // W = S*VT

    check(hipMalloc((void**)&d_A, sizeof(double) * lda * cols));
    check(hipMalloc((void**)&d_S, sizeof(double) * cols));
    check(hipMalloc((void**)&d_U, sizeof(double) * lda * rows));
    check(hipMalloc((void**)&d_VT, sizeof(double) * lda * cols));
    check(hipMalloc((void**)&devInfo, sizeof(int)));
    check(hipMalloc((void**)&d_W, sizeof(double) * lda * cols));

    check(hipMemcpy(d_A, A.data(), sizeof(double) * lda * cols, hipMemcpyHostToDevice));

    // step 3: query working space of SVD
    int lwork = 0;
    check(hipsolverDnDgesvd_bufferSize(cusolver_handle, rows, cols, &lwork));

    double* d_work = nullptr;
    check(hipMalloc((void**)&d_work, sizeof(double) * lwork));

    // step 4: compute SVD
    signed char jobu = 'A';  // all m columns of U
    signed char jobvt = 'A'; // all n columns of VT

    double* d_rwork = nullptr;

    check(hipsolverDnDgesvd(cusolver_handle,
                           jobu,
                           jobvt,
                           rows,
                           cols,
                           d_A,
                           lda,
                           d_S,
                           d_U,
                           lda, // ldu
                           d_VT,
                           lda, // ldvt,
                           d_work,
                           lwork,
                           d_rwork,
                           devInfo));

    check(hipDeviceSynchronize());

    // m-by-m unitary matrix
    std::vector<double> U(lda * rows);
    // n-by-n unitary matrix
    std::vector<double> VT(lda * cols);
    // singular value
    std::vector<double> S(cols);

    check(hipMemcpy(U.data(), d_U, sizeof(double) * lda * rows, hipMemcpyDeviceToHost));
    check(hipMemcpy(VT.data(), d_VT, sizeof(double) * lda * cols, hipMemcpyDeviceToHost));
    check(hipMemcpy(S.data(), d_S, sizeof(double) * cols, hipMemcpyDeviceToHost));
    check(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    std::cout << "after gesvd: info_gpu = " << info_gpu << "\n";

    assert(info_gpu == 0);

    std::cout << "=====\n";

    std::cout << "S = (matlab base-1)\n";
    printMatrix(cols, 1, S.data(), lda, "S");
    std::cout << "=====\n";

    std::cout << "U = (matlab base-1)\n";
    printMatrix(rows, rows, U.data(), lda, "U");
    std::cout << "=====\n";

    std::cout << "VT = (matlab base-1)\n";
    printMatrix(cols, cols, VT.data(), lda, "VT");
    std::cout << "=====\n";

    // step 5: measure error of singular value
    std::vector<double> const S_exact = {7.065283497082729, 1.040081297712078};

    double ds_sup = 0.0;
    for (int j = 0; j < cols; j++)
    {
        ds_sup = std::max(ds_sup, std::abs(S[j] - S_exact[j]));
    }

    std::cout << "|S - S_exact| = " << ds_sup << "\n";

    // step 6: |A - U*S*VT|
    // W = S*VT
    check(hipblasDdgmm(cublas_handle, HIPBLAS_SIDE_LEFT, cols, cols, d_VT, lda, d_S, 1, d_W, lda));

    // A := -U*W + A
    check(hipMemcpy(d_A, A.data(), sizeof(double) * lda * cols, hipMemcpyHostToDevice));

    double constexpr h_one = 1;
    double constexpr h_minus_one = -1;

    check(hipblasDgemm(cublas_handle,
                         HIPBLAS_OP_N,  // U
                         HIPBLAS_OP_N,  // W
                         rows,         // number of rows of A
                         cols,         // number of columns of A
                         cols,         // number of columns of U
                         &h_minus_one, // host pointer
                         d_U,          // U
                         lda,          // Leading dimension
                         d_W,          // W
                         lda,          // Leading dimension
                         &h_one,       // hostpointer
                         d_A,          // Device matrix
                         lda));

    double dR_fro = 0.0;
    check(hipblasDnrm2(cublas_handle, lda * cols, d_A, 1, &dR_fro));

    std::cout << "|A - U*S*VT| = " << dR_fro << "\n";

    // Deallocate memory
    if (d_A) hipFree(d_A);
    if (d_S) hipFree(d_S);
    if (d_U) hipFree(d_U);
    if (d_VT) hipFree(d_VT);
    if (devInfo) hipFree(devInfo);
    if (d_work) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);
    if (d_W) hipFree(d_W);

    // Destroy the handles
    if (cublas_handle) hipblasDestroy(cublas_handle);
    if (cusolver_handle) hipsolverDnDestroy(cusolver_handle);

    hipDeviceReset();

    return 0;
}
